#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <torch/torch.h>

#include <cuda/std/chrono>
#include <nvbench/nvbench.cuh>

#include "attention_kernel_sm80.cuh"
#include "attention_traits_sm80.h"
#include "kernels/attention/attention_params.h"

using namespace llm;

void attention_bench_sm80(nvbench::state& state) {
  // Collect CUPTI metrics
  state.collect_cupti_metrics();

  // Get the parameters
  const auto batch_size = state.get_int64("batch_size");
  const auto q_len = state.get_int64("q_len");
  const auto kv_len = state.get_int64("kv_len");
  const auto n_heads = state.get_int64("n_heads");
  const auto n_kv_heads = state.get_int64("n_kv_heads");
  const auto head_dim = state.get_int64("head_dim");
  const float logits_soft_cap = state.get_float64("logits_soft_cap");

  const auto options = torch::dtype(torch::kHalf).device(torch::kCUDA);
  const auto query =
      torch::randn({batch_size, n_heads, q_len, head_dim}, options);
  const auto key =
      torch::randn({batch_size, n_kv_heads, kv_len, head_dim}, options);
  const auto value =
      torch::randn({batch_size, n_kv_heads, kv_len, head_dim}, options);

  auto out = torch::empty_like(query);

  const float sm_scale = 1.0 / sqrt(head_dim);
  const auto h_stride = query.stride(1);
  const auto kv_h_stride = key.stride(1);

  constexpr int32_t kHeadDim = 64;
  constexpr int32_t kBlockM = 64;
  constexpr int32_t kBlockN = 64;

  // construct attention params
  AttentionParams params;
  params.q_ptr = query.const_data_ptr();
  params.q_stride =
      make_stride(query.stride(0), query.stride(1), query.stride(2));
  params.k_ptr = key.const_data_ptr();
  params.k_stride = make_stride(key.stride(0), key.stride(1), key.stride(2));
  params.v_ptr = value.const_data_ptr();
  params.v_stride =
      make_stride(value.stride(0), value.stride(1), value.stride(2));
  params.o_ptr = out.mutable_data_ptr();
  params.o_stride = make_stride(out.stride(0), out.stride(1), out.stride(2));
  params.alibi_slopes_ptr = nullptr;
  params.n_heads = n_heads;
  params.n_kv_heads = n_kv_heads;
  params.q_len = q_len;
  params.kv_len = kv_len;
  params.head_dim = head_dim;
  params.sm_scale = sm_scale;
  params.logits_soft_cap = logits_soft_cap;
  params.sliding_window = -1;

  using AttentionTraits = AttentionTraitsSM80<cute::half_t,
                                              kHeadDim,
                                              kBlockM,
                                              kBlockN,
                                              /*Alibi=*/false>;

  dim3 block = AttentionTraits::kThreadNum;
  dim3 grid((q_len + kBlockM - 1) / kBlockM, batch_size, n_heads);

  const auto smem_size = AttentionTraits::kSmemSize;
  auto attention_kernel = mha_kernel_sm80<AttentionTraits, AttentionParams>;

  state.exec([&](nvbench::launch& launch) {
    hipFuncSetAttribute(reinterpret_cast<const void*>(attention_kernel),
                         hipFuncAttributeMaxDynamicSharedMemorySize,
                         smem_size);
    attention_kernel<<<grid, block, smem_size, launch.get_stream()>>>(params);
  });
}

NVBENCH_BENCH(attention_bench_sm80)
    .add_int64_axis("batch_size", {1})
    .add_int64_axis("q_len", {1024})
    .add_int64_axis("kv_len", {1024})
    .add_int64_axis("n_heads", {8})
    .add_int64_axis("n_kv_heads", {8})
    .add_int64_axis("head_dim", {64})
    .add_float64_axis("logits_soft_cap", {0.0});
