#include "hip/hip_runtime.h"
#include <ATen/cuda/Exceptions.h>
#include <gtest/gtest.h>
#include <torch/torch.h>

#include "attention_kernel_sm80.cuh"
#include "attention_traits_sm80.h"

namespace llm {
namespace {
// Multi-head attention implementation using pytorch
torch::Tensor attention_ref(
    torch::Tensor query,  // [batch_size, n_heads, q_len, head_dim]
    torch::Tensor key,    // [batch_size, n_kv_heads, kv_len, head_dim]
    torch::Tensor value,  // [batch_size, n_kv_heads, kv_len, head_dim]
    torch::optional<torch::Tensor> alibi_slopes,  //[n_heads]
    float logits_soft_cap,
    int32_t sliding_window) {
  const auto q_len = query.size(2);
  const auto kv_len = key.size(2);
  const auto n_heads = query.size(1);
  const auto n_kv_heads = key.size(1);
  const auto head_dim = query.size(3);
  assert(n_heads == n_kv_heads);

  const float sm_scale = 1.0 / sqrt(head_dim);
  // query * key => [n_heads, q_seq_len, seq_len]
  auto scores = torch::einsum("bhqd,bhkd->bhqk",
                              {query.to(torch::kFloat), key.to(torch::kFloat)});
  // apply scale
  scores *= sm_scale;

  // apply softcap if needed
  if (logits_soft_cap != 0.0) {
    scores = torch::tanh(scores / logits_soft_cap) * logits_soft_cap;
  }

  // apply alibi bias
  if (alibi_slopes) {
    const auto& slopes = alibi_slopes.value();
    // calculate alibi attention bias
    // since it's causal mask, we can just use [0, 1, ...,, kv_len)
    auto distance = torch::arange(0, kv_len, query.options());
    // [n_heads, 1, kv_len]
    auto bias = distance.view({1, 1, kv_len}) * slopes.view({n_heads, 1, 1});
    scores += bias;
  }

  auto mask = torch::ones({q_len, kv_len}, torch::kBool);
  if (sliding_window >= 0) {
    // sliding window mask
    // returns the upper triangular part of a matrix
    mask = torch::triu(mask, /*diagonal=*/kv_len - q_len - sliding_window);
  }

  // apply causal mask
  // causal mask: returns the lower triangular part of a matrix
  mask = torch::tril(mask, /*diagonal=*/kv_len - q_len).to(query);
  scores = scores.masked_fill(mask == 0, -INFINITY);

  // safe softmax
  scores = torch::softmax(scores, /*dim=*/-1);

  // score * value => [batch_size, n_heads, q_seq_len, head_dim]
  return torch::einsum("bhqk,bhkd->bhqd", {scores, value.to(torch::kFloat)})
      .type_as(query);
}

torch::Tensor attention_sm80(
    torch::Tensor query,  // [batch_size, n_heads, q_len, head_dim]
    torch::Tensor key,    // [batch_size, n_kv_heads, kv_len, head_dim]
    torch::Tensor value,  // [batch_size, n_kv_heads, kv_len, head_dim]
    torch::optional<torch::Tensor> alibi_slopes,  //[n_heads]
    float logits_soft_cap,
    int32_t sliding_window) {
  const auto batch_size = query.size(0);
  const auto n_heads = query.size(1);
  const auto q_len = query.size(2);
  const auto kv_len = key.size(2);
  const auto head_dim = query.size(3);

  const auto h_stride = query.stride(1);
  const auto kv_h_stride = key.stride(1);

  auto out = torch::empty_like(query);

  // TODO: pass in alibi slope

  constexpr int32_t kHeadDim = 64;
  constexpr int32_t kBlockM = 64;
  constexpr int32_t kBlockN = 64;

  const float sm_scale = 1.0 / sqrt(head_dim);

  using AttentionTraits =
      AttentionTraitsSM80<cute::half_t, kHeadDim, kBlockM, kBlockN>;

  dim3 block = AttentionTraits::kThreadNum;
  dim3 grid((q_len + kBlockM - 1) / kBlockM, n_heads, batch_size);

  const auto smem_size = AttentionTraits::kSmemSize;
  auto attention_kernel = mha_kernel_sm80<AttentionTraits, /*Alibi=*/false>;
  C10_CUDA_CHECK(
      hipFuncSetAttribute(reinterpret_cast<const void*>(attention_kernel),
                           hipFuncAttributeMaxDynamicSharedMemorySize,
                           smem_size));
  attention_kernel<<<grid, block, smem_size>>>(out.mutable_data_ptr(),
                                               query.const_data_ptr(),
                                               key.const_data_ptr(),
                                               value.const_data_ptr(),
                                               h_stride,
                                               kv_h_stride,
                                               q_len,
                                               kv_len,
                                               sm_scale,
                                               logits_soft_cap,
                                               /*left_window_size=*/-1,
                                               /*alibi_slope=*/0.0f);
  C10_CUDA_KERNEL_LAUNCH_CHECK();
  return out;
}

}  // namespace

class AttentionKernelTest
    : public ::testing::TestWithParam<std::tuple<int64_t /*batch_size*/,
                                                 int64_t /*q_len*/,
                                                 int64_t /*kv_len*/,
                                                 int64_t /*n_heads*/,
                                                 int64_t /*n_kv_heads*/,
                                                 int64_t /*head_dim*/,
                                                 float /*logits_soft_cap*/,
                                                 bool /*alibi*/,
                                                 int32_t /*sliding_window*/>> {
 public:
  void SetUp() override {
    // Set random seed for test stability
    torch::manual_seed(0);
  }
};

TEST_P(AttentionKernelTest, MHA) {
  const auto [batch_size,
              q_len,
              kv_len,
              n_heads,
              n_kv_heads,
              head_dim,
              logits_soft_cap,
              alibi,
              sliding_window] = GetParam();

  const auto options = torch::dtype(torch::kHalf).device(torch::kCUDA);

  const auto query =
      torch::randn({batch_size, n_heads, q_len, head_dim}, options);
  const auto key =
      torch::randn({batch_size, n_kv_heads, kv_len, head_dim}, options);
  const auto value =
      torch::randn({batch_size, n_kv_heads, kv_len, head_dim}, options);

  torch::optional<torch::Tensor> alibi_slopes;
  if (alibi) {
    alibi_slopes = torch::rand(
        {n_heads}, torch::dtype(torch::kFloat32).device(torch::kCUDA));
  }

  auto ref_out = attention_ref(
      query, key, value, alibi_slopes, logits_soft_cap, sliding_window);
  auto out = attention_sm80(
      query, key, value, alibi_slopes, logits_soft_cap, sliding_window);

  EXPECT_TRUE(torch::allclose(out, ref_out, /*rtol=*/1e-3, /*atol=*/1e-3));
}

INSTANTIATE_TEST_SUITE_P(
    MHA,
    AttentionKernelTest,
    ::testing::Combine(::testing::Values(1, 2, 4),         // batch_size
                       ::testing::Values(64, 128),         // q_len
                       ::testing::Values(128, 256, 1024),  // kv_len
                       ::testing::Values(16),              // n_heads
                       ::testing::Values(16),              // n_kv_heads
                       ::testing::Values(64),              // head_dim
                       ::testing::Values(0.0, 50.0),       // logits_soft_cap
                       ::testing::Values(false),           // alibi slope
                       ::testing::Values(-1, 10)           // sliding window
                       ));

}  // namespace llm