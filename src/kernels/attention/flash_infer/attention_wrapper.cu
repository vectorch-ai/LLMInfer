//  Adapted from https://github.com/flashinfer-ai/flashinfer/
#include <c10/cuda/CUDAStream.h>
#include <torch/torch.h>

#include <cstdint>
#include <flashinfer/attention/logits_post_hook.cuh>
#include <flashinfer/attention/mask.cuh>
#include <flashinfer/attention/warp_layout.cuh>

#include "attention_wrapper.h"
#include "kv_cache.h"
#include "static_switch.h"

namespace flashinfer {

template <WarpLayout WARP_LAYOUT,
          uint32_t HEAD_DIM,
          LogitsPostHook LOGITS_POST_HOOK,
          PosEncodingMode pos_encoding_mode,
          bool ALLOW_FP16_QK_REDUCTION,
          MaskMode MASK_MODE,
          typename DTypeQ,
          typename DTypeKV,
          typename DTypeOut,
          typename IdType>
hipError_t mha_varlen_dispatch(DTypeQ* q,
                                IdType* request_indices,
                                IdType* q_tile_indices,
                                IdType* kv_tile_indices,
                                IdType* q_indptr,
                                IdType* kv_indptr,
                                paged_kv_t<DTypeKV, IdType> paged_kv,
                                uint8_t* custom_mask,
                                IdType* qk_indptr,
                                IdType* o_indptr,
                                DTypeOut* o,
                                DTypeOut* tmp_v,
                                float* tmp_s,
                                float* lse,
                                IdType* merge_indptr,
                                bool* block_valid_mask,
                                IdType* kv_chunk_size_ptr,
                                uint32_t total_num_rows,
                                uint32_t num_qo_heads,
                                uint32_t num_kv_heads,
                                uint32_t padded_batch_size,
                                int32_t window_left,
                                float logits_soft_cap,
                                float sm_scale,
                                float* alibi_slopes,
                                hipStream_t stream);

template <uint32_t HEAD_DIM,
          LogitsPostHook LOGITS_POST_HOOK,
          PosEncodingMode POS_ENCODING_MODE,
          MaskMode MASK_MODE,
          typename DTypeQ,
          typename DTypeKV,
          typename DTypeOut,
          typename IdType>
hipError_t mha_varlen_wrapper_dispatch(BatchPrefillHandler* handler,
                                        DTypeQ* q,
                                        IdType* q_indptr,
                                        IdType* kv_indptr,
                                        paged_kv_t<DTypeKV, IdType> paged_kv,
                                        uint8_t* custom_mask,
                                        IdType* qk_indptr,
                                        DTypeOut* o,
                                        float* lse,
                                        uint32_t num_qo_heads,
                                        uint32_t num_kv_heads,
                                        int32_t window_left,
                                        float logits_soft_cap,
                                        float sm_scale,
                                        float* alibi_slopes,
                                        hipStream_t stream) {
  DTypeOut* tmp_v = nullptr;
  float* tmp_s = nullptr;
  IdType *request_indices = nullptr, *qo_tile_indices = nullptr,
         *kv_tile_indices = nullptr, *o_indptr = nullptr,
         *merge_indptr = nullptr, *kv_chunk_size_ptr = nullptr;
  bool* block_valid_mask = nullptr;
  WarpLayout warp_layout;
  uint32_t padded_batch_size = 0U;
  uint32_t total_num_rows = 0U;
  tmp_v = handler->GetTempV<DTypeOut>();
  tmp_s = handler->GetTempS();
  request_indices = handler->GetRequestIndices<IdType>();
  qo_tile_indices = handler->GetQOTileIndices<IdType>();
  kv_tile_indices = handler->GetKVTileIndices<IdType>();
  block_valid_mask = handler->GetBlockValidMask();
  o_indptr = handler->GetOIndptr<IdType>();
  merge_indptr = handler->GetMergeIndptr<IdType>();
  kv_chunk_size_ptr = handler->GetKVChunkSizePtr<IdType>();
  warp_layout = handler->GetWarpLayout();
  padded_batch_size = handler->GetPaddedBatchSize();
  total_num_rows = handler->GetTotalNumRows();

  DISPATCH_WARP_LAYOUT(warp_layout, WARP_LAYOUT, {
    return mha_varlen_dispatch<WARP_LAYOUT,
                               HEAD_DIM,
                               LOGITS_POST_HOOK,
                               POS_ENCODING_MODE,
                               /*ALLOW_FP16_QK_REDUCTION=*/false,
                               MASK_MODE,
                               DTypeQ,
                               DTypeKV,
                               DTypeOut,
                               IdType>(q,
                                       request_indices,
                                       qo_tile_indices,
                                       kv_tile_indices,
                                       q_indptr,
                                       kv_indptr,
                                       paged_kv,
                                       custom_mask,
                                       qk_indptr,
                                       o_indptr,
                                       o,
                                       tmp_v,
                                       tmp_s,
                                       lse,
                                       merge_indptr,
                                       block_valid_mask,
                                       kv_chunk_size_ptr,
                                       total_num_rows,
                                       num_qo_heads,
                                       num_kv_heads,
                                       padded_batch_size,
                                       window_left,
                                       logits_soft_cap,
                                       sm_scale,
                                       alibi_slopes,
                                       stream);
  });
  return hipSuccess;
}

void BatchPrefillWrapper::Plan(torch::Tensor float_workspace_buffer,
                               torch::Tensor int_workspace_buffer,
                               torch::Tensor qo_indptr,
                               torch::Tensor paged_kv_indptr,
                               unsigned int batch_size,
                               unsigned int num_qo_heads,
                               unsigned int num_kv_heads,
                               unsigned int head_dim,
                               unsigned int page_size,
                               torch::Tensor empty_q_data) {
  CHECK_INPUT(float_workspace_buffer);
  CHECK_INPUT(int_workspace_buffer);
  // NOTE(Zihao): not necessary to be a CUDA tensor
  CHECK_CONTIGUOUS(qo_indptr);
  CHECK_CONTIGUOUS(paged_kv_indptr);
  CHECK_GQA_HEAD_DIVISIBLE(num_qo_heads, num_kv_heads);
  CHECK_DIM(1, qo_indptr);
  CHECK_DIM(1, paged_kv_indptr);
  CHECK_DIM(1, float_workspace_buffer);
  CHECK_DIM(1, int_workspace_buffer);
  CHECK_EQ(qo_indptr.size(0), batch_size + 1);
  CHECK_EQ(paged_kv_indptr.size(0), batch_size + 1);
  qo_indptr = qo_indptr.to(torch::dtype(torch::kInt32).device(torch::kCPU));
  paged_kv_indptr =
      paged_kv_indptr.to(torch::dtype(torch::kInt32).device(torch::kCPU));
  auto device = float_workspace_buffer.device();
  size_t float_workspace_size_in_bytes =
      float_workspace_buffer.size(0) * float_workspace_buffer.element_size();
  size_t int_workspace_size_in_bytes =
      int_workspace_buffer.size(0) * int_workspace_buffer.element_size();
  hipStream_t torch_current_stream =
      c10::cuda::getCurrentCUDAStream(device.index());
  handler_->SetCUDAStream(torch_current_stream);

  DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FP16(empty_q_data.scalar_type(), q_type, [&] {
    hipError_t status = handler_->Plan<q_type, int32_t>(
        static_cast<void*>(float_workspace_buffer.data_ptr()),
        float_workspace_size_in_bytes,
        static_cast<void*>(int_workspace_buffer.data_ptr()),
        int_workspace_size_in_bytes,
        static_cast<int32_t*>(qo_indptr.data_ptr()),
        static_cast<int32_t*>(paged_kv_indptr.data_ptr()),
        batch_size,
        num_qo_heads,
        num_kv_heads,
        head_dim,
        page_size);
    TORCH_CHECK(status == hipSuccess,
                "BatchPrefillWithPagedKVCache failed with error ",
                hipGetErrorString(status));
    return true;
  });
}

void BatchPrefillWrapper::UpdatePageLockedBufferSize(
    unsigned int int_workspace_size_in_bytes) {
  handler_->UpdatePageLockedBufferSize(int_workspace_size_in_bytes);
}

torch::Tensor BatchPrefillWrapper::Run(
    torch::Tensor q,
    torch::Tensor qo_indptr,
    torch::Tensor kv_indptr,
    std::optional<torch::Tensor> paged_k_cache,
    std::optional<torch::Tensor> paged_v_cache,
    torch::Tensor paged_kv_indptr,
    torch::Tensor paged_kv_indices,
    int window_left,
    float logits_soft_cap,
    float sm_scale,
    std::optional<torch::Tensor> alibi_slopes) {
  CHECK_INPUT(q);
  CHECK_INPUT(qo_indptr);
  CHECK_INPUT(kv_indptr);
  CHECK_INPUT(paged_k_cache.value());
  CHECK_INPUT(paged_v_cache.value());
  CHECK_INPUT(paged_kv_indptr);
  CHECK_INPUT(paged_kv_indices);
  auto device = q.device();
  CHECK_EQ(device, qo_indptr.device());
  CHECK_EQ(device, kv_indptr.device());
  CHECK_EQ(device, paged_k_cache->device());
  CHECK_EQ(device, paged_v_cache->device());
  CHECK_EQ(device, paged_kv_indptr.device());
  CHECK_EQ(device, paged_kv_indices.device());
  CHECK_DIM(3, q);          // (nnz_qo, H_qo, D)
  CHECK_DIM(1, qo_indptr);  // (B + 1,)

  // [max_num_pages, num_kv_heads, page_size, head_dim] for HND
  // [max_num_pages, page_size, num_kv_heads, head_dim] for HND
  CHECK_DIM(4, paged_k_cache.value());
  CHECK_DIM(4, paged_v_cache.value());

  CHECK_DIM(1, paged_kv_indptr);   // (B + 1,)
  CHECK_DIM(1, paged_kv_indices);  // (nnz_kv,)
  int64_t batch_size = qo_indptr.size(0) - 1;
  int64_t nnz_qo = q.size(0);
  int64_t num_qo_heads = q.size(1);
  int64_t head_dim = q.size(2);
  int64_t num_kv_heads, page_size;

  CHECK_EQ(paged_k_cache->size(3), head_dim);
  CHECK_EQ(paged_v_cache->size(3), head_dim);
  page_size = paged_k_cache->size(1);
  num_kv_heads = paged_k_cache->size(2);

  CHECK_GQA_HEAD_DIVISIBLE(num_qo_heads, num_kv_heads);
  CHECK_GE(qo_indptr.size(0), batch_size + 1);
  CHECK_GE(kv_indptr.size(0), batch_size + 1);
  CHECK_GE(paged_kv_indptr.size(0), batch_size + 1);
  qo_indptr = qo_indptr.to(torch::kInt32);
  kv_indptr = kv_indptr.to(torch::kInt32);
  paged_kv_indptr = paged_kv_indptr.to(torch::kInt32);
  paged_kv_indices = paged_kv_indices.to(torch::kInt32);

  hipStream_t torch_current_stream =
      c10::cuda::getCurrentCUDAStream(device.index());
  torch::Tensor o = torch::empty_like(q, q.options());
  MaskMode mask_mode = MaskMode::kCausal;
  TORCH_CHECK(logits_soft_cap >= 0.f, "logits_soft_cap must be non-negative");
  const LogitsPostHook logits_post_hook =
      logits_soft_cap > 0.f ? LogitsPostHook::kSoftCap : LogitsPostHook::kNone;
  const auto pos_encoding_mode = alibi_slopes.has_value()
                                     ? PosEncodingMode::kALiBi
                                     : PosEncodingMode::kNone;

  auto q_scalar_type = q.scalar_type();
  auto kv_scalar_type = paged_k_cache->scalar_type();

  if (q_scalar_type == kv_scalar_type) {
    DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FP16(q_scalar_type, c_type, [&] {
      return DISPATCH_logits_post_hook(logits_post_hook, LOGITS_POST_HOOK, [&] {
        paged_kv_t<c_type, int32_t> paged_kv(
            num_kv_heads,
            page_size,
            head_dim,
            batch_size,
            static_cast<c_type*>(paged_k_cache->data_ptr()),
            static_cast<c_type*>(paged_v_cache->data_ptr()),
            paged_kv_indices.data_ptr<int32_t>(),
            paged_kv_indptr.data_ptr<int32_t>());
        return DISPATCH_head_dim(head_dim, HEAD_DIM, [&] {
          return DISPATCH_mask_mode(mask_mode, MASK_MODE, [&] {
            return DISPATCH_pos_encoding_mode(
                PosEncodingMode(pos_encoding_mode), POS_ENCODING_MODE, [&] {
                  hipError_t status =
                      mha_varlen_wrapper_dispatch<HEAD_DIM,
                                                  LOGITS_POST_HOOK,
                                                  POS_ENCODING_MODE,
                                                  MASK_MODE,
                                                  c_type,
                                                  c_type,
                                                  c_type,
                                                  int32_t>(
                          handler_.get(),
                          static_cast<c_type*>(q.data_ptr()),
                          qo_indptr.data_ptr<int32_t>(),
                          kv_indptr.data_ptr<int32_t>(),
                          paged_kv,
                          /*custom_mask=*/nullptr,
                          /*qk_indptr=*/nullptr,
                          static_cast<c_type*>(o.data_ptr()),
                          /*lse=*/nullptr,
                          num_qo_heads,
                          num_kv_heads,
                          window_left,
                          logits_soft_cap,
                          sm_scale,
                          alibi_slopes.has_value()
                              ? alibi_slopes->data_ptr<float>()
                              : nullptr,
                          /*stream=*/torch_current_stream);
                  TORCH_CHECK(status == hipSuccess,
                              "BatchPrefillWithPagedKVCache failed with "
                              "error code ",
                              hipGetErrorString(status));
                  return true;
                });
          });
        });
      });
    });
  } else {
    DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FP16(q_scalar_type, q_type, [&] {
      return DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FP8(kv_scalar_type, kv_type, [&] {
        return DISPATCH_logits_post_hook(
            logits_post_hook, LOGITS_POST_HOOK, [&] {
              paged_kv_t<kv_type, int32_t> paged_kv(
                  num_kv_heads,
                  page_size,
                  head_dim,
                  batch_size,
                  static_cast<kv_type*>(paged_k_cache->data_ptr()),
                  static_cast<kv_type*>(paged_v_cache->data_ptr()),
                  paged_kv_indices.data_ptr<int32_t>(),
                  paged_kv_indptr.data_ptr<int32_t>());
              return DISPATCH_head_dim(head_dim, HEAD_DIM, [&] {
                return DISPATCH_mask_mode(mask_mode, MASK_MODE, [&] {
                  return DISPATCH_pos_encoding_mode(
                      PosEncodingMode(pos_encoding_mode),
                      POS_ENCODING_MODE,
                      [&] {
                        hipError_t status =
                            mha_varlen_wrapper_dispatch<HEAD_DIM,
                                                        LOGITS_POST_HOOK,
                                                        POS_ENCODING_MODE,
                                                        MASK_MODE,
                                                        q_type,
                                                        kv_type,
                                                        q_type,
                                                        int32_t>(
                                handler_.get(),
                                static_cast<q_type*>(q.data_ptr()),
                                qo_indptr.data_ptr<int32_t>(),
                                kv_indptr.data_ptr<int32_t>(),
                                paged_kv,
                                /*custom_mask=*/nullptr,
                                /*qk_indptr=*/nullptr,
                                static_cast<q_type*>(o.data_ptr()),
                                /*lse=*/nullptr,
                                num_qo_heads,
                                num_kv_heads,
                                window_left,
                                logits_soft_cap,
                                sm_scale,
                                alibi_slopes.has_value()
                                    ? alibi_slopes->data_ptr<float>()
                                    : nullptr,
                                /*stream=*/torch_current_stream);
                        TORCH_CHECK(status == hipSuccess,
                                    "BatchPrefillWithPagedKVCache failed "
                                    "with error code ",
                                    hipGetErrorString(status));
                        return true;
                      });
                });
              });
            });
      });
    });
  }

  return o;
}

}  // namespace flashinfer