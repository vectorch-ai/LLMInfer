#include "hip/hip_runtime.h"
#include <ATen/cuda/Exceptions.h>
#include <absl/random/random.h>
#include <gtest/gtest.h>
#include <torch/torch.h>

#include "attention_kernel_sm80.cuh"
#include "attention_traits_sm80.h"
#include "cute/layout.hpp"
#include "kernels/attention/attention_params.h"

namespace llm {
namespace {
// Multi-head attention implementation using pytorch
torch::Tensor attention_ref(
    torch::Tensor query,  // [batch_size, n_heads, q_len, head_dim]
    torch::Tensor key,    // [batch_size, n_kv_heads, kv_len, head_dim]
    torch::Tensor value,  // [batch_size, n_kv_heads, kv_len, head_dim]
    torch::optional<torch::Tensor> alibi_slopes,  //[n_heads]
    float logits_soft_cap,
    int32_t sliding_window) {
  const auto q_len = query.size(2);
  const auto kv_len = key.size(2);
  const auto n_heads = query.size(1);
  const auto n_kv_heads = key.size(1);
  const auto head_dim = query.size(3);
  assert(kv_len >= q_len);

  if (n_heads != n_kv_heads) {
    assert(n_heads % n_kv_heads == 0);
    const auto group_size = n_heads / n_kv_heads;
    key = key.repeat_interleave(/*repeats=*/group_size, /*dim=*/-3);
    value = value.repeat_interleave(/*repeats=*/group_size, /*dim=*/-3);
  }

  const float sm_scale = 1.0 / sqrt(head_dim);
  // query * key => [n_heads, q_seq_len, seq_len]
  auto scores = torch::einsum("bhqd,bhkd->bhqk",
                              {query.to(torch::kFloat), key.to(torch::kFloat)});
  // apply scale
  scores *= sm_scale;

  // apply softcap if needed
  if (logits_soft_cap != 0.0) {
    scores = torch::tanh(scores / logits_soft_cap) * logits_soft_cap;
  }

  // apply alibi bias
  if (alibi_slopes) {
    const auto& slopes = alibi_slopes.value();
    // calculate alibi attention bias
    // since it's causal mask, we can just use [0, 1, ...,, kv_len)
    auto distance = torch::arange(0, kv_len, query.options());
    // [n_heads, 1, kv_len]
    auto bias = distance.view({1, 1, kv_len}) * slopes.view({n_heads, 1, 1});
    scores += bias;
  }

  auto mask = torch::ones({q_len, kv_len}, torch::kBool);
  if (sliding_window >= 0) {
    // sliding window mask
    // returns the upper triangular part of a matrix
    mask = torch::triu(mask, /*diagonal=*/kv_len - q_len - sliding_window);
  }

  // apply causal mask
  // causal mask: returns the lower triangular part of a matrix
  mask = torch::tril(mask, /*diagonal=*/kv_len - q_len).to(query);
  scores = scores.masked_fill(mask == 0, -INFINITY);

  // safe softmax
  scores = torch::softmax(scores, /*dim=*/-1);

  // score * value => [batch_size, n_heads, q_seq_len, head_dim]
  return torch::einsum("bhqk,bhkd->bhqd", {scores, value.to(torch::kFloat)})
      .type_as(query);
}

torch::Tensor attention_varlen_sm80(
    torch::Tensor query,          // [n_heads, q_seq_len, head_dim]
    torch::Tensor key,            // [n_kv_heads, kv_seq_len, head_dim]
    torch::Tensor value,          // [n_kv_heads, kv_seq_len, head_dim]
    torch::Tensor k_cu_seq_lens,  // [batch_size+1]
    torch::Tensor q_cu_seq_lens,  // [batch_size+1]
    torch::optional<torch::Tensor> alibi_slopes,  //[n_heads]
    float logits_soft_cap,
    int32_t sliding_window) {
  const auto n_heads = query.size(0);
  const auto n_kv_heads = key.size(0);
  const auto q_len = query.size(1);
  const auto kv_len = key.size(1);
  const auto head_dim = query.size(2);
  const auto batch_size = q_cu_seq_lens.size(0) - 1;

  auto out = torch::empty_like(query);

  // TODO: pass in alibi slope

  constexpr int32_t kHeadDim = 64;
  constexpr int32_t kBlockM = 64;
  constexpr int32_t kBlockN = 64;

  const float sm_scale = 1.0 / sqrt(head_dim);

  // construct attention params
  VarLenAttentionParams params;
  params.q_ptr = query.const_data_ptr();
  params.q_stride = make_stride(query.stride(0), query.stride(1));
  params.k_ptr = key.const_data_ptr();
  params.k_stride = make_stride(key.stride(0), key.stride(1));
  params.v_ptr = value.const_data_ptr();
  params.v_stride = make_stride(value.stride(0), value.stride(1));
  params.o_ptr = out.mutable_data_ptr();
  params.o_stride = make_stride(out.stride(0), out.stride(1));
  params.alibi_slopes_ptr = alibi_slopes.has_value()
                                ? alibi_slopes.value().const_data_ptr<float>()
                                : nullptr;
  params.n_heads = n_heads;
  params.n_kv_heads = n_kv_heads;
  params.head_dim = head_dim;
  params.sm_scale = sm_scale;
  params.logits_soft_cap = logits_soft_cap;
  params.sliding_window = sliding_window;

  params.cu_seqlens_q = q_cu_seq_lens.const_data_ptr<int32_t>();
  params.cu_seqlens_kv = k_cu_seq_lens.const_data_ptr<int32_t>();

  if (alibi_slopes.has_value()) {
    using AttentionTraits = AttentionTraitsSM80<cute::half_t,
                                                kHeadDim,
                                                kBlockM,
                                                kBlockN,
                                                /*Alibi=*/true>;

    dim3 block = AttentionTraits::kThreadNum;
    dim3 grid((q_len + kBlockM - 1) / kBlockM, batch_size, n_heads);

    const auto smem_size = AttentionTraits::kSmemSize;
    auto attention_kernel =
        mha_kernel_sm80<AttentionTraits, VarLenAttentionParams>;
    C10_CUDA_CHECK(
        hipFuncSetAttribute(reinterpret_cast<const void*>(attention_kernel),
                             hipFuncAttributeMaxDynamicSharedMemorySize,
                             smem_size));
    attention_kernel<<<grid, block, smem_size>>>(params);
  } else {
    using AttentionTraits = AttentionTraitsSM80<cute::half_t,
                                                kHeadDim,
                                                kBlockM,
                                                kBlockN,
                                                /*Alibi=*/false>;

    dim3 block = AttentionTraits::kThreadNum;
    dim3 grid((q_len + kBlockM - 1) / kBlockM, batch_size, n_heads);

    const auto smem_size = AttentionTraits::kSmemSize;
    auto attention_kernel =
        mha_kernel_sm80<AttentionTraits, VarLenAttentionParams>;
    C10_CUDA_CHECK(
        hipFuncSetAttribute(reinterpret_cast<const void*>(attention_kernel),
                             hipFuncAttributeMaxDynamicSharedMemorySize,
                             smem_size));
    attention_kernel<<<grid, block, smem_size>>>(params);
  }
  C10_CUDA_KERNEL_LAUNCH_CHECK();
  return out;
}

}  // namespace

class AttentionKernelVarlenTest
    : public ::testing::TestWithParam<std::tuple<int64_t /*batch_size*/,
                                                 int64_t /*q_len*/,
                                                 int64_t /*kv_len*/,
                                                 int64_t /*n_heads*/,
                                                 int64_t /*n_kv_heads*/,
                                                 int64_t /*head_dim*/,
                                                 float /*logits_soft_cap*/,
                                                 bool /*alibi*/,
                                                 int32_t /*sliding_window*/>> {
 public:
  void SetUp() override {
    // Set random seed for test stability
    torch::manual_seed(0);
  }
};

TEST_P(AttentionKernelVarlenTest, VarLen) {
  const auto [batch_size,
              max_q_len,
              max_kv_len,
              n_heads,
              n_kv_heads,
              head_dim,
              logits_soft_cap,
              alibi,
              sliding_window] = GetParam();

  const auto options = torch::dtype(torch::kHalf).device(torch::kCUDA);

  // random generate seq lens with size in [1, max_seq_len]
  std::vector<int32_t> q_cu_seq_lens_vec = {0};
  std::vector<int32_t> k_cu_seq_lens_vec = {0};
  int32_t n_kv_tokens = 0;
  int32_t n_q_tokens = 0;
  absl::BitGen gen;
  for (int i = 0; i < batch_size; ++i) {
    // q_len: [1, q_max_seq_len]
    // const int32_t q_len =
    //     absl::Uniform<int>(absl::IntervalClosedClosed, gen, 1, max_q_len);
    const int32_t q_len = max_q_len;
    n_q_tokens += q_len;
    q_cu_seq_lens_vec.push_back(n_q_tokens);

    // kv_len >= q_len
    int32_t kv_len = max_kv_len;
    // if (q_len < max_kv_len) {
    //   // sample kv_len from [q_len, kv_max_seq_len]
    //   kv_len = absl::Uniform<int>(
    //       absl::IntervalClosedClosed, gen, q_len, max_kv_len);
    // }
    n_kv_tokens += kv_len;
    k_cu_seq_lens_vec.push_back(n_kv_tokens);
    assert(kv_len >= q_len);
  }

  // construct non-contiguous query, key and value
  // generate query, key and value
  torch::Tensor query = torch::rand({n_heads, n_q_tokens, head_dim}, options);
  torch::Tensor key = torch::rand({n_kv_heads, n_kv_tokens, head_dim}, options);
  torch::Tensor value =
      torch::rand({n_kv_heads, n_kv_tokens, head_dim}, options);

  torch::Tensor q_cu_seq_lens = torch::tensor(
      q_cu_seq_lens_vec, torch::dtype(torch::kInt32).device(torch::kCUDA));
  torch::Tensor k_cu_seq_lens = torch::tensor(
      k_cu_seq_lens_vec, torch::dtype(torch::kInt32).device(torch::kCUDA));

  torch::optional<torch::Tensor> alibi_slopes;
  if (alibi) {
    alibi_slopes = torch::rand(
        {n_heads}, torch::dtype(torch::kFloat32).device(torch::kCUDA));
  }

  // auto ref_out = attention_ref(
  //     query, key, value, alibi_slopes, logits_soft_cap, sliding_window);
  auto out = attention_varlen_sm80(query,
                                   key,
                                   value,
                                   q_cu_seq_lens,
                                   k_cu_seq_lens,
                                   alibi_slopes,
                                   logits_soft_cap,
                                   sliding_window);

  // EXPECT_TRUE(torch::allclose(out, ref_out, /*rtol=*/1e-3, /*atol=*/1e-3));
}

INSTANTIATE_TEST_SUITE_P(
    VarLen,
    AttentionKernelVarlenTest,
    ::testing::Combine(::testing::Values(10),      // batch_size
                       ::testing::Values(100),      // max_q_len
                       ::testing::Values(1000),      // max_kv_len
                       ::testing::Values(16),      // n_heads
                       ::testing::Values(16),      // n_kv_heads
                       ::testing::Values(64),     // head_dim
                       ::testing::Values(0.0),    // logits_soft_cap
                       ::testing::Values(false),  // alibi slope
                       ::testing::Values(-1)      // sliding window
                       ));

// ::testing::Combine(
// ::testing::Values(1, 2, 4),                          // batch_size
// ::testing::Values(1, 62, 125),                       // max_q_len
// ::testing::Values(127, 287, 1000),                   // max_kv_len
// ::testing::Values(6),                                // n_heads
// ::testing::Values(6 /*mha*/, 3 /*gqa*/, 1 /*mqa*/),  // n_kv_heads
// ::testing::Values(64),                               // head_dim
// ::testing::Values(0.0, 50.0),                        // logits_soft_cap
// ::testing::Values(false, true),                      // alibi slope
// ::testing::Values(-1, 0, 10)                         // sliding window
// ));

}  // namespace llm