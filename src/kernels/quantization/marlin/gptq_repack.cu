#include "hip/hip_runtime.h"
// Adapted from: https://github.com/vllm-project/vllm

#include <c10/cuda/CUDAStream.h>
#include <glog/logging.h>
#include <torch/torch.h>

#include <cstdint>

namespace marlin {
namespace {
// Repack params
static constexpr int repack_stages = 8;

static constexpr int repack_threads = 256;

static constexpr int tile_size = 16;

static constexpr int tile_k_size = tile_size;
static constexpr int tile_n_size = tile_k_size * 4;
constexpr int div_ceil(int a, int b) { return (a + b - 1) / b; }

__device__ inline void cp_async_fence() {
  asm volatile("cp.async.commit_group;\n" ::);
}

template <int n>
__device__ inline void cp_async_wait() {
  asm volatile("cp.async.wait_group %0;\n" ::"n"(n));
}

__device__ inline void cp_async4(void* smem_ptr, const void* glob_ptr) {
  const int BYTES = 16;
  uint32_t smem = static_cast<uint32_t>(__cvta_generic_to_shared(smem_ptr));
  asm volatile(
      "{\n"
      "   cp.async.cg.shared.global [%0], [%1], %2;\n"
      "}\n" ::"r"(smem),
      "l"(glob_ptr),
      "n"(BYTES));
}

template <int const num_threads, int const num_bits, bool const has_perm>
__global__ void gptq_marlin_repack_kernel(
    uint32_t const* __restrict__ b_q_weight_ptr,
    uint32_t const* __restrict__ perm_ptr,
    uint32_t* __restrict__ out_ptr,
    int size_k,
    int size_n) {
  constexpr int pack_factor = 32 / num_bits;

  int k_tiles = size_k / tile_k_size;
  int n_tiles = size_n / tile_n_size;
  int block_k_tiles = div_ceil(k_tiles, gridDim.x);

  int start_k_tile = blockIdx.x * block_k_tiles;
  if (start_k_tile >= k_tiles) {
    return;
  }

  int finish_k_tile = min(start_k_tile + block_k_tiles, k_tiles);

  // Wait until the next thread tile has been loaded to shared memory.
  auto wait_for_stage = [&]() {
    // We only have `stages - 2` active fetches since we are double buffering
    // and can only issue the next fetch when it is guaranteed that the previous
    // shared memory load is fully complete (as it may otherwise be
    // overwritten).
    cp_async_wait<repack_stages - 2>();
    __syncthreads();
  };

  extern __shared__ int4 sh[];

  // perm_size = 16 / 4
  constexpr int perm_size = tile_k_size / 4;

  int4* sh_perm_ptr = sh;
  int4* sh_pipe_ptr = sh_perm_ptr;
  if constexpr (has_perm) {
    sh_pipe_ptr += perm_size;
  }

  constexpr int tile_ints = tile_k_size / pack_factor;

  constexpr int stage_n_threads = tile_n_size / 4;
  constexpr int stage_k_threads = has_perm ? tile_k_size : tile_ints;
  constexpr int stage_size = stage_k_threads * stage_n_threads;

  auto load_perm_to_shared = [&](int k_tile_id) {
    int first_k_int4 = (k_tile_id * tile_k_size) / 4;

    int4 const* perm_int4_ptr = reinterpret_cast<int4 const*>(perm_ptr);

    if (threadIdx.x < perm_size) {
      sh_perm_ptr[threadIdx.x] = perm_int4_ptr[first_k_int4 + threadIdx.x];
    }
    __syncthreads();
  };

  auto fetch_to_shared = [&](int pipe, int k_tile_id, int n_tile_id) {
    if (n_tile_id >= n_tiles) {
      cp_async_fence();
      return;
    }

    int first_n = n_tile_id * tile_n_size;

    int4* sh_ptr = sh_pipe_ptr + stage_size * pipe;

    if constexpr (has_perm) {
      if (threadIdx.x < stage_size) {
        int k_id = threadIdx.x / stage_n_threads;
        int n_id = threadIdx.x % stage_n_threads;

        uint32_t const* sh_perm_int_ptr =
            reinterpret_cast<uint32_t const*>(sh_perm_ptr);

        int src_k = sh_perm_int_ptr[k_id];
        int src_k_packed = src_k / pack_factor;

        cp_async4(
            &sh_ptr[k_id * stage_n_threads + n_id],
            reinterpret_cast<int4 const*>(&(
                b_q_weight_ptr[src_k_packed * size_n + first_n + (n_id * 4)])));
      }

    } else {
      if (threadIdx.x < stage_size) {
        int k_id = threadIdx.x / stage_n_threads;
        int n_id = threadIdx.x % stage_n_threads;

        int first_k = k_tile_id * tile_k_size;
        int first_k_packed = first_k / pack_factor;

        cp_async4(&sh_ptr[k_id * stage_n_threads + n_id],
                  reinterpret_cast<int4 const*>(
                      &(b_q_weight_ptr[(first_k_packed + k_id) * size_n +
                                       first_n + (n_id * 4)])));
      }
    }

    cp_async_fence();
  };

  auto repack_tile = [&](int pipe, int k_tile_id, int n_tile_id) {
    if (n_tile_id >= n_tiles) {
      return;
    }

    int warp_id = threadIdx.x / 32;
    int th_id = threadIdx.x % 32;

    if (warp_id >= 4) {
      return;
    }

    int tc_col = th_id / 4;
    int tc_row = (th_id % 4) * 2;

    constexpr int tc_offsets[4] = {0, 1, 8, 9};

    int cur_n = warp_id * 16 + tc_col;

    constexpr int sh_stride = 64;
    constexpr uint32_t mask = (1 << num_bits) - 1;

    int4* sh_stage_ptr = sh_pipe_ptr + stage_size * pipe;
    uint32_t* sh_stage_int_ptr = reinterpret_cast<uint32_t*>(sh_stage_ptr);

    uint32_t* sh_perm_int_ptr = reinterpret_cast<uint32_t*>(sh_perm_ptr);

    uint32_t vals[8];

    if constexpr (has_perm) {
      for (int i = 0; i < 4; i++) {
        int k_idx = tc_row + tc_offsets[i];

        uint32_t src_k = sh_perm_int_ptr[k_idx];
        uint32_t src_k_pos = src_k % pack_factor;

        uint32_t b1_val = sh_stage_int_ptr[k_idx * sh_stride + cur_n];
        uint32_t b1_cur_val = (b1_val >> (src_k_pos * num_bits)) & mask;

        uint32_t b2_val = sh_stage_int_ptr[k_idx * sh_stride + cur_n + 8];
        uint32_t b2_cur_val = (b2_val >> (src_k_pos * num_bits)) & mask;

        vals[i] = b1_cur_val;
        vals[4 + i] = b2_cur_val;
      }

    } else {
      uint32_t b1_vals[tile_ints];
      uint32_t b2_vals[tile_ints];

#pragma unroll
      for (int i = 0; i < tile_ints; i++) {
        b1_vals[i] = sh_stage_int_ptr[cur_n + sh_stride * i];
        b2_vals[i] = sh_stage_int_ptr[cur_n + 8 + sh_stride * i];
      }

#pragma unroll
      for (int i = 0; i < 4; i++) {
        int cur_elem = tc_row + tc_offsets[i];
        int cur_int = cur_elem / pack_factor;
        int cur_pos = cur_elem % pack_factor;

        vals[i] = (b1_vals[cur_int] >> (cur_pos * num_bits)) & mask;
        vals[4 + i] = (b2_vals[cur_int] >> (cur_pos * num_bits)) & mask;
      }
    }

    constexpr int tile_size = tile_k_size * tile_n_size / pack_factor;
    int out_offset = (k_tile_id * n_tiles + n_tile_id) * tile_size;

    // Result of:
    // https://github.com/NVIDIA/FasterTransformer/blob/main/src/fastertransformer/cutlass_extensions/include/cutlass_extensions/interleaved_numeric_conversion.h
    if constexpr (num_bits == 4) {
      constexpr int pack_idx[8] = {0, 2, 4, 6, 1, 3, 5, 7};

      uint32_t res = 0;
#pragma unroll
      for (int i = 0; i < 8; i++) {
        res |= vals[pack_idx[i]] << (i * 4);
      }

      out_ptr[out_offset + th_id * 4 + warp_id] = res;

    } else {
      constexpr int pack_idx[4] = {0, 2, 1, 3};

      uint32_t res1 = 0;
      uint32_t res2 = 0;
#pragma unroll
      for (int i = 0; i < 4; i++) {
        res1 |= vals[pack_idx[i]] << (i * 8);
        res2 |= vals[4 + pack_idx[i]] << (i * 8);
      }

      out_ptr[out_offset + th_id * 8 + (warp_id * 2) + 0] = res1;
      out_ptr[out_offset + th_id * 8 + (warp_id * 2) + 1] = res2;
    }
  };

  auto start_pipes = [&](int k_tile_id, int n_tile_id) {
#pragma unroll
    for (int pipe = 0; pipe < repack_stages - 1; pipe++) {
      fetch_to_shared(pipe, k_tile_id, n_tile_id + pipe);
    }

    wait_for_stage();
  };
#pragma unroll
  for (int k_tile_id = start_k_tile; k_tile_id < finish_k_tile; k_tile_id++) {
    int n_tile_id = 0;

    if constexpr (has_perm) {
      load_perm_to_shared(k_tile_id);
    }

    start_pipes(k_tile_id, n_tile_id);

    while (n_tile_id < n_tiles) {
#pragma unroll
      for (int pipe = 0; pipe < repack_stages; pipe++) {
        fetch_to_shared((pipe + repack_stages - 1) % repack_stages,
                        k_tile_id,
                        n_tile_id + pipe + repack_stages - 1);
        repack_tile(pipe, k_tile_id, n_tile_id + pipe);
        wait_for_stage();
      }
      n_tile_id += repack_stages;
    }
  }
}

#define CALL_IF(NUM_BITS, HAS_PERM)                                           \
  else if (num_bits == NUM_BITS && has_perm == HAS_PERM) {                    \
    hipFuncSetAttribute(reinterpret_cast<const void*>(                                                     \
        marlin::gptq_marlin_repack_kernel<marlin::repack_threads),             \
                                          NUM_BITS,                           \
                                          HAS_PERM>,                          \
        hipFuncAttributeMaxDynamicSharedMemorySize,                          \
        max_shared_mem);                                                      \
    marlin::                                                                  \
        gptq_marlin_repack_kernel<marlin::repack_threads, NUM_BITS, HAS_PERM> \
        <<<blocks, marlin::repack_threads, max_shared_mem, stream>>>(         \
            b_q_weight_ptr, perm_ptr, out_ptr, size_k, size_n);               \
  }

}  // namespace

void gptq_repack(const torch::Tensor& b_q_weight,  // (k/pack_factor, n)
                 const torch::Tensor& perm,        // ?
                 torch::Tensor& out,               // (k/16, n*16/pack_factor)
                 int64_t num_bits) {
  CHECK(num_bits == 4 || num_bits == 8);

  const int64_t pack_factor = 32 / num_bits;
  const int64_t size_k = b_q_weight.size(0) * pack_factor;
  const int64_t size_n = b_q_weight.size(1);

  // Verify compatibility with marlin tile of 16x64
  CHECK(size_k % 16 == 0);
  CHECK(size_n % 64 == 0);

  // Verify device and strides
  CHECK(b_q_weight.device().is_cuda());
  CHECK(b_q_weight.is_contiguous());
  CHECK(b_q_weight.dtype() == torch::kInt);
  CHECK(perm.device().is_cuda());
  CHECK(perm.is_contiguous());
  CHECK(perm.dtype() == torch::kInt);

  // Detect if there is act_order
  bool has_perm = perm.defined();

  // Get ptrs
  const uint32_t* b_q_weight_ptr =
      reinterpret_cast<const uint32_t*>(b_q_weight.data_ptr());
  const uint32_t* perm_ptr = reinterpret_cast<const uint32_t*>(perm.data_ptr());
  uint32_t* out_ptr = reinterpret_cast<uint32_t*>(out.data_ptr());

  // Get dev info
  int dev = b_q_weight.get_device();
  hipStream_t stream = at::cuda::getCurrentCUDAStream(dev);
  int blocks = 0;
  hipDeviceGetAttribute(&blocks, hipDeviceAttributeMultiprocessorCount, dev);

  int max_shared_mem = 0;
  hipDeviceGetAttribute(
      &max_shared_mem, hipDeviceAttributeSharedMemPerBlockOptin, dev);
  TORCH_CHECK(max_shared_mem > 0);

  // NOLINTNEXTLINE
  if (false) {
  }
  CALL_IF(4, false)
  CALL_IF(4, true)
  CALL_IF(8, false)
  CALL_IF(8, true)
  else {
    LOG(FATAL) << "Unsupported repack config: num_bits = " << num_bits
               << ", has_perm = " << has_perm;
  }
}

}  // namespace marlin