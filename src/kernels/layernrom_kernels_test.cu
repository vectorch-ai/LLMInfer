#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>

#include <cstdio>

#include "layernorm_kernels.h"

template <typename T>
void printMatrix(T* a, int m, int n) {
  for (int i = 0; i < m; i++) {
    for (int j = 0; j < n; j++) {
      printf("%f ", (float)a[i * n + j]);
    }
    puts("");
  }
  puts("");
}

template <>
void printMatrix<half2>(half2* a, int m, int n) {
  for (int i = 0; i < m; i++) {
    for (int j = 0; j < n; j++) {
      printf(
          "%f %f ", __half2float(a[i * n + j].x), __half2float(a[i * n + j].y));
    }
    puts("");
  }
  puts("");
}

void layernorm_kernel_half2_test() {
  float epsilon = 1e-6;
  int m = 2;
  int n = 2;

  half2* out = (half2*)malloc(m * n * sizeof(half2));
  half2* input = (half2*)malloc(m * n * sizeof(half2));
  half2* weight = (half2*)malloc(m * n * sizeof(half2));
  half2* bias = (half2*)malloc(m * n * sizeof(half2));

  for (int i = 0; i < m; i++) {
    for (int j = 0; j < n; j++) {
      input[i * n + j] = half2(__float2half((float)(i * n + j * 2)),
                               __float2half((float)(i * n + j * 2 + 1)));
      weight[i * n + j] = half2(__float2half(1.), __float2half(1.));
      bias[i * n + j] = half2(__float2half(0.), __float2half(0.));
    }
  }

  half2* dout;
  half2* dinput;
  half2* dweight;
  half2* dbias;
  hipMalloc((void**)&dout, sizeof(half2) * m * n);
  hipMalloc((void**)&dinput, sizeof(half2) * m * n);
  hipMalloc((void**)&dweight, sizeof(half2) * m * n);
  hipMalloc((void**)&dbias, sizeof(half2) * m * n);

  hipMemcpy(dinput, input, sizeof(half2) * m * n, hipMemcpyHostToDevice);
  hipMemcpy(dweight, weight, sizeof(half2) * m * n, hipMemcpyHostToDevice);
  hipMemcpy(dbias, bias, sizeof(half2) * m * n, hipMemcpyHostToDevice);

  llm::kernel::invoke_layernorm_kernel<half2>(
      dout, dinput, dweight, dbias, epsilon, m, n);

  hipMemcpy(out, dout, sizeof(half2) * m * n, hipMemcpyDeviceToHost);

  printf("---------- test half2 layernorm kernel -----------\n");
  printf("input:\n");
  printMatrix<half2>(input, m, n);
  printf("weights:\n");
  printMatrix<half2>(weight, m, n);
  printf("bias:\n");
  printMatrix<half2>(bias, m, n);
  printf("outputs:\n");
  printMatrix<half2>(out, m, n);
}

void layernorm_kernel_float_test() {
  float epsilon = 1e-6;
  int m = 2;
  int n = 4;

  float* out = (float*)malloc(m * n * sizeof(float));
  float* input = (float*)malloc(m * n * sizeof(float));
  float* weight = (float*)malloc(m * n * sizeof(float));
  float* bias = (float*)malloc(m * n * sizeof(float));

  for (int i = 0; i < m; i++) {
    for (int j = 0; j < n; j++) {
      input[i * n + j] = (float)(i * n + j);
      weight[i * n + j] = 1.;
      bias[i * n + j] = 0.;
    }
  }

  float* dout;
  float* dinput;
  float* dweight;
  float* dbias;
  hipMalloc((void**)&dout, sizeof(float) * m * n);
  hipMalloc((void**)&dinput, sizeof(float) * m * n);
  hipMalloc((void**)&dweight, sizeof(float) * m * n);
  hipMalloc((void**)&dbias, sizeof(float) * m * n);

  hipMemcpy(dinput, input, sizeof(float) * m * n, hipMemcpyHostToDevice);
  hipMemcpy(dweight, weight, sizeof(float) * m * n, hipMemcpyHostToDevice);
  hipMemcpy(dbias, bias, sizeof(float) * m * n, hipMemcpyHostToDevice);

  llm::kernel::invoke_layernorm_kernel<float>(
      dout, dinput, dweight, dbias, epsilon, m, n);

  hipMemcpy(out, dout, sizeof(float) * m * n, hipMemcpyDeviceToHost);

  printf("---------- test float layernorm kernel -----------\n");
  printf("input:\n");
  printMatrix<float>(input, m, n);
  printf("weights:\n");
  printMatrix<float>(weight, m, n);
  printf("bias:\n");
  printMatrix<float>(bias, m, n);
  printf("outputs:\n");
  printMatrix<float>(out, m, n);
}

int main() {
  layernorm_kernel_float_test();
  layernorm_kernel_half2_test();
  return 0;
}