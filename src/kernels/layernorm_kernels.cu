#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <torch/torch.h>

#include "dispatch.h"
#include "layernorm_kernels.h"
#include "reduce_kernel_utils.cuh"

namespace llm::kernel {

// calculate the root mean square norm.
// equation: x -> w * x / sqrt(E[x^2] + eps)
// The mean is calculated over the last dimension
// equilvalent to layernorm module in the T5 style No bias and no subtraction of
// mean.
template <typename T>
__global__ void rms_norm_kernel(T* __restrict__ out,
                                const T* __restrict__ input,
                                const T* __restrict__ weight,
                                const float epsilon,
                                int64_t n) {
  const auto tidx = threadIdx.x;
  const auto bidx = blockIdx.x;

  __shared__ float s_variance;
  float variance = 0.0f;

  for (int64_t i = tidx; i < n; i += blockDim.x) {
    const float x = input[bidx * n + i];
    variance += x * x;
  }
  variance = block_reduce_sum<float>(variance);
  if (tidx == 0) {
    s_variance = rsqrtf(variance / n + epsilon);
  }
  __syncthreads();

  for (int64_t i = tidx; i < n; i += blockDim.x) {
    const int64_t idx = bidx * n + i;
    const float x = input[idx];
    out[idx] = (T)(x * s_variance) * weight[i];
  }
}

void rms_norm(torch::Tensor& out,
              torch::Tensor input,
              torch::Tensor weight,
              float epsilon) {
  DCHECK(input.is_contiguous()) << "input tensor must be contiguous";
  DCHECK(out.is_contiguous()) << "output tensor must be contiguous";

  const int64_t n = input.size(1);

  dim3 grid(input.size(0));
  dim3 block(std::min<int>(n, 1024));
  DISPATCH_FLOATING_TYPES(input.scalar_type(), "rms_norm_kernel", [&] {
    rms_norm_kernel<scalar_t>
        <<<grid, block, 0, at::cuda::getCurrentCUDAStream()>>>(
            out.data_ptr<scalar_t>(),
            input.data_ptr<scalar_t>(),
            weight.data_ptr<scalar_t>(),
            epsilon,
            n);
  });
}

// calculate the root mean square norm.
// equation: x -> w * x / sqrt(E[x^2] + eps)
// The mean is calculated over the last dimension
// equilvalent to layernorm module in the T5 style No bias and no subtraction of
// mean.
template <typename T>
__global__ void rms_norm_residual_kernel(T* __restrict__ out,
                                         T* __restrict__ residual,
                                         const T* __restrict__ input,
                                         const T* __restrict__ weight,
                                         const float epsilon,
                                         int64_t n) {
  const auto tidx = threadIdx.x;
  const auto bidx = blockIdx.x;

  __shared__ float s_variance;
  float variance = 0.0f;

  for (int64_t i = tidx; i < n; i += blockDim.x) {
    const int64_t idx = bidx * n + i;
    const float r = residual[idx];
    const float x = r + input[idx];
    residual[idx] = x;
    variance += x * x;
  }
  variance = block_reduce_sum<float>(variance);
  if (tidx == 0) {
    s_variance = rsqrtf(variance / n + epsilon);
  }
  __syncthreads();

  for (int64_t i = tidx; i < n; i += blockDim.x) {
    const int64_t idx = bidx * n + i;
    const float x = residual[idx];
    out[idx] = (T)(x * s_variance) * weight[i];
  }
}

void rms_norm_residual(torch::Tensor& out,
                       torch::Tensor& residual,
                       torch::Tensor input,
                       torch::Tensor weight,
                       float epsilon) {
  DCHECK(input.is_contiguous()) << "input tensor must be contiguous";
  DCHECK(out.is_contiguous()) << "output tensor must be contiguous";
  DCHECK(residual.is_contiguous()) << "residual tensor must be contiguous";

  const int64_t n = input.size(1);

  dim3 grid(input.size(0));
  dim3 block(std::min<int>(n, 1024));
  DISPATCH_FLOATING_TYPES(input.scalar_type(), "rms_norm_residual_kernel", [&] {
    rms_norm_residual_kernel<scalar_t>
        <<<grid, block, 0, at::cuda::getCurrentCUDAStream()>>>(
            out.data_ptr<scalar_t>(),
            residual.data_ptr<scalar_t>(),
            input.data_ptr<scalar_t>(),
            weight.data_ptr<scalar_t>(),
            epsilon,
            n);
  });
}

// equation: x -> (x - E[x]) / sqrt(Var[x] + eps) * w + b
// The mean and standard-deviation are calculated over the last dimension
template <typename T>
__global__ void layer_norm_kernel(T* __restrict__ out,
                                  const T* __restrict__ input,
                                  const T* __restrict__ weight,
                                  const T* __restrict__ bias,
                                  const float epsilon,
                                  int64_t n) {
  const auto tidx = threadIdx.x;
  const auto bidx = blockIdx.x;

  __shared__ float s_mean;
  __shared__ float s_variance;
  float mean = 0.0f;
  float variance = 0.0f;

  // calculate mean of the input.
  for (int64_t i = tidx; i < n; i += blockDim.x) {
    mean += input[bidx * n + i];
  }
  mean = block_reduce_sum<float>(mean);
  if (tidx == 0) {
    s_mean = mean / n;
  }
  __syncthreads();

  // calculate variance of the input.
  for (int64_t i = tidx; i < n; i += blockDim.x) {
    const float x = input[bidx * n + i] - s_mean;
    variance += x * x;
  }
  variance = block_reduce_sum<float>(variance);
  if (tidx == 0) {
    s_variance = rsqrtf(variance / n + epsilon);
  }
  __syncthreads();

  for (int64_t i = tidx; i < n; i += blockDim.x) {
    const int64_t idx = bidx * n + i;
    float local_out = (input[idx] - s_mean) * s_variance * weight[i];
    if (bias != nullptr) {
      local_out += bias[i];
    }
    out[idx] = (T)(local_out);
  }
}

// equation: x -> (x - E[x]) / sqrt(Var[x] + eps) * w + b
// The mean and standard-deviation are calculated over the last dimension
template <>
__global__ void layer_norm_kernel<half2>(half2* __restrict__ out,
                                         const half2* __restrict__ input,
                                         const half2* __restrict__ weight,
                                         const half2* __restrict__ bias,
                                         const float epsilon,
                                         int64_t n) {
  const int tidx = threadIdx.x;
  const int bidx = blockIdx.x;

  __shared__ half s_mean;
  __shared__ half s_variance;
  half2 mean = make_half2(__float2half(0.0f), __float2half(0.0f));
  half2 variance = make_half2(__float2half(0.0f), __float2half(0.0f));

  // calculate mean of the input.
  for (int i = tidx; i < n; i += blockDim.x) {
    const int idx = bidx * n + i;
    mean = __hadd2(mean, __ldg(&input[idx]));
  }
  mean = block_reduce_sum<half2>(mean);
  if (tidx == 0) {
    s_mean = __hdiv(__hadd(mean.x, mean.y), __float2half((float)n * 2));
  }
  __syncthreads();

  // calculate variance of the input.
  for (int i = tidx; i < n; i += blockDim.x) {
    const half2 x = __hsub2(input[bidx * n + i], make_half2(s_mean, s_mean));
    variance = __hadd2(variance, __hmul2(x, x));
  }
  variance = block_reduce_sum<half2>(variance);
  if (tidx == 0) {
    s_variance = __hadd(variance.x, variance.y);
    s_variance = __hdiv(s_variance, __float2half((float)n * 2));
    s_variance = __hadd(s_variance, __float2half(epsilon));
    s_variance = hrsqrt(s_variance);
  }
  __syncthreads();

  for (int i = tidx; i < n; i += blockDim.x) {
    const int idx = bidx * n + i;
    half2 local_out = __ldg(&input[idx]);
    local_out = __hsub2(local_out, make_half2(s_mean, s_mean));
    local_out = __hmul2(local_out, make_half2(s_variance, s_variance));
    local_out = __hmul2(local_out, __ldg(&weight[i]));
    if (bias != nullptr) {
      local_out = __hadd2(local_out, __ldg(&bias[i]));
    }
    out[idx] = local_out;
  }
}

void layer_norm(torch::Tensor& out,
                torch::Tensor input,
                torch::Tensor weight,
                torch::Tensor bias,
                float epsilon) {
  DCHECK(input.is_contiguous()) << "input tensor must be contiguous";
  DCHECK(out.is_contiguous()) << "output tensor must be contiguous";

  const int64_t n = input.size(1);

  dim3 grid(input.size(0));
  dim3 block(std::min<int>(n, 1024));
  DISPATCH_FLOATING_TYPES(input.scalar_type(), "layer_norm_kernel", [&] {
    layer_norm_kernel<scalar_t>
        <<<grid, block, 0, at::cuda::getCurrentCUDAStream()>>>(
            out.data_ptr<scalar_t>(),
            input.data_ptr<scalar_t>(),
            weight.data_ptr<scalar_t>(),
            bias.defined() ? bias.data_ptr<scalar_t>() : nullptr,
            epsilon,
            n);
  });
}

template <typename T>
void invoke_layernorm_kernel(T* out,
                             const T* input,
                             const T* weight,
                             const T* bias,
                             const float epsilon,
                             int m,
                             int n) {
  layer_norm_kernel<T><<<m, n>>>(out, input, weight, bias, epsilon, n);
}

template <>
void invoke_layernorm_kernel<half2>(half2* out,
                                    const half2* input,
                                    const half2* weight,
                                    const half2* bias,
                                    const float epsilon,
                                    int m,
                                    int n) {
  layer_norm_kernel<half2><<<m, n>>>(out, input, weight, bias, epsilon, n);
}
template <>
void invoke_layernorm_kernel<float>(float* out,
                                    const float* input,
                                    const float* weight,
                                    const float* bias,
                                    const float epsilon,
                                    int m,
                                    int n) {
  layer_norm_kernel<float><<<m, n>>>(out, input, weight, bias, epsilon, n);
}

template <>
void invoke_layernorm_kernel<half>(half* out,
                                   const half* input,
                                   const half* weight,
                                   const half* bias,
                                   const float epsilon,
                                   int m,
                                   int n) {
  int half_n = n / 2;
  half2* out_ptr = (half2*)out;
  const half2* input_ptr = (const half2*)input;
  const half2* weight_ptr = (const half2*)weight;
  const half2* bias_ptr = (const half2*)bias;

  dim3 block(std::min(half_n, 1024));
  layer_norm_kernel<half2>
      <<<m, block>>>(out_ptr, input_ptr, weight_ptr, bias_ptr, epsilon, half_n);
}
}  // namespace llm::kernel