#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <torch/torch.h>

#include "dispatch.h"
#include "kv_cache_kernels.h"
namespace llm::kernel {

template <typename T>
__global__ void set_kv_cache_kernel(
    const int* __restrict__ slot_ids,  // [n_tokens]
    const T* __restrict__ keys,        // [n_tokens, n_heads, head_dim]
    const T* __restrict__ values,      // [n_tokens, n_heads, head_dim]
    T* __restrict__ key_cache,
    T* __restrict__ value_cache,
    int64_t k_stride,
    int64_t v_stride,
    int64_t n_kv_heads,
    int64_t head_dim,
    int64_t block_size) {
  // block/token index
  const int64_t bid = blockIdx.x;
  // which slot to write to
  const int64_t slot_id = slot_ids[bid];
  // block index
  const int64_t block_idx = slot_id / block_size;
  // offset within block
  const int64_t block_offset = slot_id % block_size;

  // base index for the block in cache
  const int64_t block_base_idx = block_idx * block_size * n_kv_heads * head_dim;

  // copy value one by one for the token
  for (int64_t i = threadIdx.x; i < n_kv_heads * head_dim; i += blockDim.x) {
    const int64_t k_src_idx = bid * k_stride + i;
    const int64_t v_src_idx = bid * v_stride + i;

    // cache: [n_blocks, block_size, n_heads, head_dim]
    const int64_t head_base_idx =
        block_base_idx + block_offset * n_kv_heads * head_dim;

    // which head to write to
    const int head_idx = i / head_dim;
    // which dim within head to write to
    const int head_offset = i % head_dim;
    const int64_t dst_idx = head_base_idx + head_idx * head_dim + head_offset;

    key_cache[dst_idx] = keys[k_src_idx];
    value_cache[dst_idx] = values[v_src_idx];
  }
}

void set_kv_cache(
    const torch::Tensor& slot_ids,  // [n_tokens]
    const torch::Tensor& keys,      // [n_tokens, n_kv_heads, head_dim]
    const torch::Tensor& values,    // [n_tokens, n_kv_heads, head_dim]
    torch::Tensor& key_cache,       // [n_blocks, block_size, n_heads, head_dim]
    torch::Tensor& value_cache) {
  // keys and values should be continuous at n_kv_heads and head_dim dims
  CHECK(keys.stride(-1) == 1 && keys.stride(-2) == keys.size(-1));
  CHECK(values.stride(-1) == 1 && values.stride(-2) == values.size(-1));

  const int64_t n_tokens = keys.size(-3);
  const int64_t n_kv_heads = keys.size(-2);
  const int64_t head_dim = keys.size(-1);
  const int64_t block_size = key_cache.size(-3);
  // it is possible that keys and values have different strides
  const int64_t k_stride = keys.stride(-3);
  const int64_t v_stride = values.stride(-3);
  const int64_t n = n_kv_heads * head_dim;

  dim3 grid(n_tokens);
  dim3 block(std::min<int>(n, 1024));
  DISPATCH_FLOATING_TYPES(keys.scalar_type(), "set_kv_cache_kernel", [&] {
    set_kv_cache_kernel<scalar_t>
        <<<grid, block, 0, at::cuda::getCurrentCUDAStream()>>>(
            slot_ids.data_ptr<int>(),
            keys.data_ptr<scalar_t>(),
            values.data_ptr<scalar_t>(),
            key_cache.data_ptr<scalar_t>(),
            value_cache.data_ptr<scalar_t>(),
            k_stride,
            v_stride,
            n_kv_heads,
            head_dim,
            block_size);
  });
}

}  // namespace llm::kernel
